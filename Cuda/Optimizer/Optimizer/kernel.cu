
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <string>
#include "file_reader.h"

using namespace std;

int main()
{
	string path = "P:\\U\\Some2DGame\\Cuda\\info\\data.bytes";
	tuple<char*, int> blobTuple = file_reader::readFile(path);
	char* blob = get<0>(blobTuple);
	int blobLength = get<1>(blobTuple);

	int metaLength;
	memcpy(&metaLength, blob, 4);
	int combinedDataOffset = metaLength + 4;

	short spritesCount;
	memcpy(&spritesCount, blob + combinedDataOffset + 2, 2);
	short sizingsBlobLength;
	memcpy(&sizingsBlobLength, blob + combinedDataOffset + 4, 2);

	short sizingsCount = sizingsBlobLength / 4;

	int registryStructureLength = 8;

	char* sizingsBlob = blob + combinedDataOffset + 6;
	char* registryBlob = sizingsBlob + sizingsBlobLength;
	int registryBlobLength = spritesCount * registryStructureLength;
	char* dataBlob = registryBlob + registryBlobLength;
	int dataBlobLength = blobLength - registryBlobLength - sizingsBlobLength - combinedDataOffset - 6;

	char* deviceSizingsPtr;
	hipMalloc((void**)&deviceSizingsPtr, sizingsBlobLength);
	char* deviceRegistryPtr;
	hipMalloc((void**)&deviceRegistryPtr, registryBlobLength);
	char* deviceDataPtr;
	hipMalloc((void**)&deviceDataPtr, dataBlobLength);

	hipMemcpy(deviceSizingsPtr, sizingsBlob, sizingsBlobLength, hipMemcpyHostToDevice);
	hipMemcpy(deviceRegistryPtr, registryBlob, sizingsBlobLength, hipMemcpyHostToDevice);
	hipMemcpy(deviceDataPtr, dataBlob, sizingsBlobLength, hipMemcpyHostToDevice);


	hipFree(deviceSizingsPtr);
	hipFree(deviceRegistryPtr);
	hipFree(deviceDataPtr);
	free(blob);

    return 0;
}
